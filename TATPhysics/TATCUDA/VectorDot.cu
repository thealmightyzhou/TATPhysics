#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>

const int thread_per_block = 256;

__global__ void Dot(float* a, float* b, float* c, int size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ float sh[thread_per_block];
	float temp = 0;
	while (tid < size)
	{
		temp += a[tid] * b[tid];
		tid += gridDim.x * blockDim.x;
	}

	sh[threadIdx.x] = temp;
	__syncthreads();

	int i = threadIdx.x / 2;
	while (i > 0)
	{
		if (threadIdx.x < i)
		{
			sh[threadIdx.x] += sh[threadIdx.x + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0)
	{
		c[blockIdx.x] = sh[0];
	}
}

extern "C"
float CUDADot(float* a, float* b, int size)
{
	hipError_t cudaStatus;
	int block_per_grid = (size + thread_per_block - 1) / thread_per_block;

	float* partial_c = (float*)(malloc(block_per_grid * sizeof(float)));
	
	float* dev_a, * dev_b, * dev_c, c;
	hipMalloc((void**)&dev_a, size * sizeof(float));
	hipMalloc((void**)&dev_b, size * sizeof(float));
	hipMalloc((void**)&dev_c, block_per_grid * sizeof(float));

	for (int i = 0; i < size; ++i)
	{
		a[i] = i;
		b[i] = i * 2;
	}

	hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);

	Dot << <block_per_grid, thread_per_block >> > (dev_a, dev_b, dev_c, size);

	cudaStatus = hipMemcpy(partial_c, dev_c, block_per_grid * sizeof(float), hipMemcpyDeviceToHost);
	c = 0;
	for (int i = 0; i < block_per_grid; ++i)
	{
		c += partial_c[i];
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	free(partial_c);

	return c;
}
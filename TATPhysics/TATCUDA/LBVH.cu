#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "../TATBasis/TATErrorReporter.h"
#include "../TATBroadPhase/LBVH.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <map>
#include <stack>
#include <queue>

const int thread_per_block = 256;

//====================
__global__ void DevGetMinMax(float* c, int size, float* res)
{
	__shared__ float buffer[thread_per_block * 2 * 3];

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int index = threadIdx.x;

	for (int i = 0; i < 3; ++i)
	{
		buffer[index + i * thread_per_block] = FLT_MAX;
		buffer[index + (i + 3) * thread_per_block] = -FLT_MAX;
	}

	__syncthreads();

	while (idx < size)
	{
		for (int i = 0; i < 3; ++i)
		{
			if (c[i * size + idx] < buffer[i * thread_per_block + index])
			{
				buffer[i * thread_per_block + index] = c[i * size + idx];
			}
			if (buffer[(i + 3) * thread_per_block + index] < c[i * size + idx])
			{
				buffer[(i + 3) * thread_per_block + index] = c[i * size + idx];
			}
		}

		idx += blockDim.x * gridDim.x;
	}

	__syncthreads();

	int offset = thread_per_block / 2;
	while (offset > 0)
	{
		if (index < offset)
		{
			for (int i = 0; i < 3; ++i)
			{
				if (buffer[index + offset + thread_per_block * i] < buffer[index + thread_per_block * i])
					buffer[index + thread_per_block * i] = buffer[index + offset + thread_per_block * i];

				if (buffer[index + (3 + i) * thread_per_block] < buffer[index + (3 + i) * thread_per_block + offset])
					buffer[index + (3 + i) * thread_per_block] = buffer[index + (3 + i) * thread_per_block + offset];
			}
		}

		__syncthreads();
		offset /= 2;
	}

	for (int i = 0; i < 3; ++i)
	{
		res[blockIdx.x * 6 + i] = buffer[thread_per_block * i];
		res[blockIdx.x * 6 + i + 3] = buffer[thread_per_block * (i + 3)];
	}
}

#define USE_GPUASSIGN 0

#if(USE_GPUASSIGN)
__global__ void Assign(TATVector3* buffer, int size, float* datas)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{
		for (int i = 0; i < 3; ++i)
		{
			datas[i + size * idx] = buffer[idx].m_Datas[i];
		}
	}

	__syncthreads();
}
#endif

extern "C"
void GetMinMax(TATVector3 * datas, int size, TATVector3 & min, TATVector3 & max)
{
	//x0 x1 x2 y0 y1 y2 z0 z1 z2 (3 * size)
	int block_num = (size + thread_per_block - 1) / thread_per_block;

#if(!USE_GPUASSIGN)
	float* host_buffer = (float*)malloc(3 * size * sizeof(float));
#endif
	float* res_buffer = (float*)malloc(block_num * 6 * sizeof(float));
	float* dev_buffer, * dev_res_buffer;
	hipMalloc((void**)&dev_buffer, 3 * size * sizeof(float));
	hipMalloc((void**)&dev_res_buffer, block_num * 6 * sizeof(float));

#if(USE_GPUASSIGN)
	TATVector3* dev_vec;
	hipMalloc((void**)&dev_vec, size * sizeof(TATVector3));
	hipMemcpy(dev_vec, datas, size * sizeof(TATVector3), hipMemcpyHostToDevice);
	Assign < << block_num, thread_per_block >> > (dev_vec, size, dev_buffer);
#else
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < 3; ++j)
		{
			host_buffer[i + size * j] = datas[i].m_Datas[j];
		}
	}

	//Xmin Ymin Zmin Xmax Ymax Zmax 
	hipMemcpy(dev_buffer, host_buffer, 3 * size * sizeof(float), hipMemcpyHostToDevice);
#endif

	DevGetMinMax << <block_num, thread_per_block >> > (dev_buffer, size, dev_res_buffer);

	hipMemcpy(res_buffer, dev_res_buffer, block_num * 6 * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 1; i < block_num; ++i)
	{
		for (int j = 0; j < 3; ++j)
		{
			if (res_buffer[i * 6 + j] < res_buffer[j])
				res_buffer[j] = res_buffer[i * 6 + j];
			if (res_buffer[j + 3] < res_buffer[i * 6 + j + 3])
				res_buffer[j + 3] = res_buffer[i * 6 + j + 3];
		}
	}

	memcpy(min.m_Datas, res_buffer, 3 * sizeof(float));
	memcpy(max.m_Datas, res_buffer + 3, 3 * sizeof(float));
#if(!USE_GPUASSIGN)
	free(host_buffer);
#endif
	free(res_buffer);
	hipFree(dev_buffer);
	hipFree(dev_res_buffer);
}

__device__ int Sign(int i)
{
	if (i >= 0)
		return 1;
	if (i < 0)
		return -1;
}

int Host_Sign(int i)
{
	if (i > 0)
		return 1;
	if (i < 0)
		return -1;
	return 0;
}

__device__ int Prefix(UINT i, UINT j, UINT* buffer, int max)
{
	if (j >= max || j < 0)
		return -1;

	return __clz(buffer[i] ^ buffer[j]);
}

int Host_Prefix(UINT i, UINT j, UINT* buffer, int max)
{
	if (j >= max || j < 0)
		return -1;

	UINT same = buffer[i] ^ buffer[j];

	int count = 0;
	while (same > 0)
	{
		same = same >> 1;
		count++;
	}
	return 32 - count;
}

//									 leaf      internal left    internal right
//total size 3*size-2 [0,3*size-3] [0,size-1] [size,2*size-2] [2*size-1,3*size-3]
__global__ void DevGenLBVH(UINT* buffer, bool* internal, int size)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size - 1)
	{
		int d = Sign(Prefix(i, i + 1, buffer, size) - Prefix(i, i - 1, buffer, size));
		int min_pre = Prefix(i, i - d, buffer, size);
		int max_step = 1;
		int pre;
		do
		{
			max_step *= 2;
			pre = Prefix(i, i + d * max_step, buffer, size);

		} while (pre > min_pre);

		int step = max_step / 2;
		int curr_offset = 0;

		while (step > 0)
		{
			pre = Prefix(i, i + (curr_offset + step) * d, buffer, size);
			if (pre > min_pre)
			{
				curr_offset += step;
			}

			step /= 2;
		}

		int j = i + curr_offset * d;
		int prefix_node = Prefix(i, j, buffer, size);

		//int npow = 0;
		step = 1;
		while (curr_offset > 0)
		{
			curr_offset = curr_offset >> 1;
			//npow++;
			step *= 2;
		}

		//step = (int)pow(2.0, (double)npow);

		int split = 0;

		while (step > 0)
		{
			pre = Prefix(i, i + (split + step) * d, buffer, size);
			if (pre > prefix_node)
			{
				split += step;
			}
			step /= 2;
		}

		int cut = (i + split * d + (d <= 0 ? d : 0));

		if (cut == (i < j ? i : j))
		{
			internal[i] = false;
			buffer[size + i] = cut;
		}
		else
		{
			internal[i] = true;
			buffer[size + i] = cut;
		}

		if ((cut + 1) == (i > j ? i : j))
		{
			internal[size - 1 + i] = false;
			buffer[2 * size - 1 + i] = (cut + 1);
		}
		else
		{
			internal[size - 1 + i] = true;
			buffer[2 * size - 1 + i] = (cut + 1);
		}

	}
}

void HostGenLBVH(UINT* buffer, bool* internal, int size)
{
	for (int i = 0; i < size - 1; ++i)
	{
		int d = Host_Sign(Host_Prefix(i, i + 1, buffer, size) - Host_Prefix(i, i - 1, buffer, size));
		int min_pre = Host_Prefix(i, i - d, buffer, size);
		int max_step = 1;
		int pre;
		do
		{
			max_step *= 2;
			pre = Host_Prefix(i, i + d * max_step, buffer, size);

		} while (pre > min_pre);

		int step = max_step / 2;
		int curr_offset = 0;

		while (step > 0)
		{
			pre = Host_Prefix(i, i + (curr_offset + step) * d, buffer, size);
			if (pre > min_pre)
			{
				curr_offset += step;
			}

			step /= 2;
		}

		int j = i + curr_offset * d;
		int prefix_node = Host_Prefix(i, j, buffer, size);

		int npow = 0;
		while (curr_offset > 0)
		{
			curr_offset = curr_offset >> 1;
			npow++;
		}

		step = pow(2, npow);

		int split = 0;

		while (step > 0)
		{
			pre = Host_Prefix(i, i + (split + step) * d, buffer, size);
			if (pre > prefix_node)
			{
				split += step;
			}
			step /= 2;
		} 

		int cut = (i + split * d + (d <= 0 ? d : 0));

		UINT pr = Host_Prefix(i, cut, buffer, size);
		UINT aft = Host_Prefix(i, cut + 1, buffer, size);

		if (cut == (i < j ? i : j))
		{
			internal[i] = false;
			buffer[size + i] = cut;
		}
		else
		{
			internal[i] = true;
			buffer[size + i] = cut;
		}

		if ((cut + 1) == (i > j ? i : j))
		{
			internal[size - 1 + i] = false;
			buffer[2 * size - 1 + i] = (cut + 1);
		}
		else
		{
			internal[size - 1 + i] = true;
			buffer[2 * size - 1 + i] = (cut + 1);
		}
	}
}

extern "C"
void BuildLBVH(std::vector<LBVNode>& nodes, TATVector3* pos, std::vector<LBVNode>& internal_nodes, int num)
{
	int block_num = (num + thread_per_block - 1) / thread_per_block;

	TATVector3 min, max;
	GetMinMax(pos, num, min, max);
	TATVector3 unit = max - min;

	for (int i = 0; i < num; ++i)//parallel
	{
		nodes[i].m_UnitCenter = (nodes[i].m_Center - min) / unit;
		nodes[i].GenMorton();
	}

	sort(nodes.begin(), nodes.end());

	UINT* host_buffer = (UINT*)malloc((3 * num - 2) * sizeof(UINT));

	for (int i = 0; i < num; ++i)
	{
		host_buffer[i] = nodes[i].m_MortonCode;
	}

	UINT* dev_buffer;
	hipMalloc((void**)&dev_buffer, (3 * num - 2) * sizeof(UINT));
	hipMemcpy(dev_buffer, host_buffer, (3 * num - 2) * sizeof(UINT), hipMemcpyHostToDevice);

	bool* dev_internalbuffer;
	hipMalloc((void**)&dev_internalbuffer, sizeof(bool) * (2 * num - 2));
	bool* host_internalbuffer = (bool*)malloc(sizeof(bool) * (2 * num - 2));

	//HostGenLBVH(host_buffer, host_internalbuffer, num);
	//UINT* host_bufferTemp = (UINT*)malloc((3 * num - 2) * sizeof(UINT));
	//bool* host_internalbufferTemp = (bool*)malloc(sizeof(bool) * (2 * num - 2));

	DevGenLBVH << <block_num, thread_per_block >> > (dev_buffer, dev_internalbuffer, num);
	hipDeviceSynchronize();
	hipMemcpy(host_internalbuffer, dev_internalbuffer, (2 * num - 2) * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(host_buffer, dev_buffer, (3 * num - 2) * sizeof(UINT), hipMemcpyDeviceToHost);
	//hipMemcpy(host_internalbufferTemp, dev_internalbuffer, (2 * num - 2) * sizeof(bool), hipMemcpyDeviceToHost);
	//hipMemcpy(host_bufferTemp, dev_buffer, (3 * num - 2) * sizeof(UINT), hipMemcpyDeviceToHost);

	/*for (int i = 0; i < (2 * num - 2); ++i)
	{
		if (host_internalbufferTemp[i] != host_internalbuffer[i])
		{
			int stop = 1;
		}
	}
	for (int i = 0; i < (3 * num - 2); ++i)
	{
		if (host_bufferTemp[i] != host_buffer[i])
		{
			int stop = 1;
		}
	}*/

	internal_nodes.resize(num - 1);

	LBVNode* lchild, * rchild;
	for (int i = 0; i < num - 1; ++i) //parallel
	{
		internal_nodes[i].m_IsInternal = true;

		bool linternal = host_internalbuffer[i];
		bool rinternal = host_internalbuffer[num - 1 + i];

		if (!linternal)
			internal_nodes[i].m_Children[0] = &nodes[host_buffer[num + i]];
		else
			internal_nodes[i].m_Children[0] = &internal_nodes[host_buffer[num + i]];
		if (!rinternal)
			internal_nodes[i].m_Children[1] = &nodes[host_buffer[2 * num - 1 + i]];
		else
			internal_nodes[i].m_Children[1] = &internal_nodes[host_buffer[2 * num - 1 + i]];

		lchild = internal_nodes[i].m_Children[0];
		rchild = internal_nodes[i].m_Children[1];
		lchild->m_Parent = &internal_nodes[i];
		rchild->m_Parent = &internal_nodes[i];
	}

	free(host_buffer);
	free(host_internalbuffer);
	hipFree(dev_buffer);
	hipFree(dev_internalbuffer);
}
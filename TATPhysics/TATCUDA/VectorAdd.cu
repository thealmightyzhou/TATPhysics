#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void add(int size, int* a, int* b, int* c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < size)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

extern "C"
void VectorAdd(int size, int* a, int* b,int *c)
{
	int* dev_a, * dev_b, * dev_c;

	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMalloc((void**)&dev_c, size * sizeof(int));

	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	add << <128, 128 >> > (size, dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}